
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int N = 10;
    int size = N * sizeof(int);

    // Host arrays
    int h_a[N], h_b[N], h_c[N];

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device arrays
    int *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel on the GPU with N threads
    add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Free memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
